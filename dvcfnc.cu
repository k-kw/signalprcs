#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <cmath>
#include <time.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include ""

#include "my_all.h"
#include "complex_array_class.h"
#include "dvcfnc.cuh"

//乱数ライブラリインクルード
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

typedef unsigned char uc;

using namespace std;

//CUDA
#ifndef __HIPCC__
#define __HIPCC__
#endif 

#define sqr(x) ((x)*(x))


//関数群

////template of under function
//template <class Type>
//__global__ void cusetcucomplex(hipComplex* com, Type* Re, Type* Im, int size)
//{
//
//    int idx = blockDim.x * blockIdx.x + threadIdx.x;
//
//    if (idx < size) {
//        com[idx] = make_hipComplex((float)Re[idx], (float)Im[idx]);
//    }
//}


//double to hipComplex
__global__ void cusetcucomplex(hipComplex* com, double* Re, double* Im, int size)
{

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < size) {
        com[idx] = make_hipComplex((float)Re[idx], (float)Im[idx]);
    }
}

// unsigned char to hipComplex
// num thread
__global__ void uc2cucomplex(hipComplex* com, uc* Re, int num, int size)
{

    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num) {
        for(int j=0; j<size; j++){
            com[idx*size+j] = make_hipComplex((float)Re[idx*size+j], 0.0f);
        }
    }
}



//normalization after fft 2d
__global__ void normfft(hipfftComplex* dev, int x, int y)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < x * y) {
        dev[idx] = make_hipComplex(hipCrealf(dev[idx]) / (x * y), hipCimagf(dev[idx]) / (x * y));
    }
}

// calculate power and normalization after fft 1d
// num thread
__global__ void pow_norm_fft1d(uc* pow, hipfftComplex* dev, int num, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num) {
        for(int j=0; j<size; j++){
            pow[idx*size+j] = (uc)round(sqrt(sqr(hipCrealf(dev[idx*size+j]))+sqr(hipCimagf(dev[idx*size+j])))/size);
        }
    }
}


// signal process 1d
// LPF
// 高周波領域から何％カットするか(cutrate)
// 計算自体は周波数シフトしていない状態を想定 
__global__ void LPF(hipComplex* dev, float cutrate, int num, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num) {
        int strt=(size/2)*(1-cutrate), end=(size/2)*(1+cutrate);

        for(int j=strt; j<end; j++){
            dev[idx*size+j] = make_hipComplex(0.0f, 0.0f);
        }
    }
}

// HPF
// 低周波領域から何％カットするか(cutrate)
// 計算自体は周波数シフトしていない状態を想定 
__global__ void HPF(hipComplex* dev, float cutrate, int num, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < num) {
        int end=(size/2)*cutrate;

        for(int j=0; j<end; j++){
            dev[idx*size+j] = make_hipComplex(0.0f, 0.0f);
            dev[idx*size+(size-1-j)] = make_hipComplex(0.0f, 0.0f);
        }
    }
}



//1D fft complex2complex 
// xはデータサイズ
void fft_1D_C2C(int x, hipfftComplex*dev, int batch)
{
    hipfftHandle plan;
    hipfftPlan1d(&plan, x, HIPFFT_C2C, batch);
    hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);
    hipfftDestroy(plan);
}

void ifft_1D_C2C(int x, hipfftComplex*dev, int batch)
{
    hipfftHandle plan;
    hipfftPlan1d(&plan, x, HIPFFT_C2C, batch);
    hipfftExecC2C(plan, dev, dev, HIPFFT_BACKWARD);
    hipfftDestroy(plan);
}


// R2C 使えるとメモリ食わないかも？
// //1D fft real2complex
// void fft_1D_R2C(int x, hipfftComplex*dev, int batch)
// {
//     hipfftHandle plan;
//     hipfftPlan1d(&plan, x, HIPFFT_R2C, batch);
//     hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);
//     hipfftDestroy(plan);
// }


//2d fft complex2complex
void fft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
    hipfftHandle plan;

    //hipfftPlan2d 第2引数 : 最も遅く変化する次元のサイズ
    //hipfftPlan2d 第3引数 : 最も速く変化する次元のサイズ
    hipfftPlan2d(&plan, y, x, HIPFFT_C2C);
    hipfftExecC2C(plan, dev, dev, HIPFFT_FORWARD);
    hipfftDestroy(plan);
}

//2d inverse fft complex2complex
void ifft_2D_cuda_dev(int x, int y, hipfftComplex* dev)
{
    hipfftHandle plan;

    //hipfftPlan2d 第2引数 : 最も遅く変化する次元のサイズ
    //hipfftPlan2d 第3引数 : 最も速く変化する次元のサイズ
    hipfftPlan2d(&plan, y, x, HIPFFT_C2C);
    hipfftExecC2C(plan, dev, dev, HIPFFT_BACKWARD);
    hipfftDestroy(plan);
}

//cufftcomplex to My_ComArray
void cufftcom2mycom(My_ComArray_2D* out, hipfftComplex* in, int s) {
    for (int i = 0; i < s; i++) {
        out->Re[i] = (double)hipCrealf(in[i]);
        out->Im[i] = (double)hipCimagf(in[i]);

    }
}


//make angular spectrum method's H 
__global__ void Hcudaf(float* Re, float* Im, int x, int y, float u, float v, float z, float lam)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < y && idx < x) {
        Re[idy * x + idx] = cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
        Im[idy * x + idx] = sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2))));
    }
}

//make angular spectrum method's H (hipComplex)
__global__ void HcudacuCom(hipComplex* H, int x, int y, float z, float d, float lam)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    float u = 1 / (x * d), v = 1 / (y * d);


    if (idy < y && idx < x) {
        H[idy * x + idx] = make_hipComplex(cos(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2)))),
            sin(2 * M_PI * z * sqrt(sqr(1 / lam) - sqr(u * ((float)idx - x / 2)) - sqr(v * ((float)idy - y / 2)))));
    }
}

__global__ void  shiftf(float* ore, float* oim, float* re, float* im, int x, int y)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < y && idx < x) {

        if (idx < x / 2 && idy < y / 2) {
            ore[idy * x + idx] = re[(idy + y / 2) * x + (idx + x / 2)];
            ore[(idy + y / 2) * x + (idx + x / 2)] = re[idy * x + idx];
            oim[idy * x + idx] = im[(idy + y / 2) * x + (idx + x / 2)];
            oim[(idy + y / 2) * x + (idx + x / 2)] = im[idy * x + idx];
        }
        else if (idx >= x / 2 && idy < y / 2) {
            ore[idy * x + idx] = re[(idy + y / 2) * x + (idx - x / 2)];
            ore[(idy + y / 2) * x + (idx - x / 2)] = re[idy * x + idx];
            oim[idy * x + idx] = im[(idy + y / 2) * x + (idx - x / 2)];
            oim[(idy + y / 2) * x + (idx - x / 2)] = im[idy * x + idx];
        }
    }
}

//use
__global__ void shiftCom(hipComplex* out, hipComplex* in, int x, int y)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idy < y && idx < x) {

        if (idx < x / 2 && idy < y / 2) {
            out[idy * x + idx] = in[(idy + y / 2) * x + (idx + x / 2)];
            out[(idy + y / 2) * x + (idx + x / 2)] = in[idy * x + idx];

        }
        else if (idx >= x / 2 && idy < y / 2) {
            out[idy * x + idx] = in[(idy + y / 2) * x + (idx - x / 2)];
            out[(idy + y / 2) * x + (idx - x / 2)] = in[idy * x + idx];

        }
    }
}


//floatXcufftCom
__global__ void mulcomcufftcom(hipfftComplex* out, float* re, float* im, hipfftComplex* in, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        out[idx] = make_hipComplex(re[idx] * hipCrealf(in[idx]) - im[idx] * hipCimagf(in[idx]),
            re[idx] * hipCimagf(in[idx]) + im[idx] * hipCrealf(in[idx]));

    }
}


//doubleXcufftCom
__global__ void muldoublecomcufftcom(hipfftComplex* out, double* re, double* im, hipfftComplex* in, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        out[idx] = make_hipComplex((float)re[idx] * hipCrealf(in[idx]) - (float)im[idx] * hipCimagf(in[idx]),
            (float)re[idx] * hipCimagf(in[idx]) + (float)im[idx] * hipCrealf(in[idx]));

    }
}


//use
__global__ void Cmulfft(hipComplex* out, hipComplex* fin, hipComplex* in, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    //レジスタver
    //hipComplex tmp1, tmp2;
    

    if (idx < s) {
        /*tmp1 = make_hipComplex(hipCrealf(fin[idx]), hipCimagf(fin[idx]));
        tmp2 = make_hipComplex(hipCrealf(in[idx]), hipCimagf(in[idx]));*/
        //out[idx] = hipCmulf(tmp1, tmp2);

        out[idx] = hipCmulf(fin[idx], in[idx]);

    }

}

//use
__global__ void pad_cufftcom2cufftcom(hipfftComplex* out, int lx, int ly, hipfftComplex* in, int sx, int sy)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < sx && idy < sy) {
        out[(idy + ly / 4) * lx + (idx + lx / 4)] = in[idy * sx + idx];
    }

}

__global__ void elimpad(hipfftComplex* out, int sx, int sy, hipfftComplex* in, int lx, int ly)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < sx && idy < sy) {
        out[idy * sx + idx] = in[(idy + ly / 4) * lx + (idx + lx / 4)];
    }
}

__global__ void elimpad2Cmulfft(hipComplex* outmlt, hipComplex* opponent, 
    int sx, int sy, hipComplex* in, int lx, int ly)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    hipComplex tmp1, tmp2;

    if (idx < sx && idy < sy) {
        //真ん中を取り出す
        tmp1 = make_hipComplex(hipCrealf(in[(idy + ly / 4) * lx + (idx + lx / 4)]), 
            hipCimagf(in[(idy + ly / 4) * lx + (idx + lx / 4)]));

        //レンズ配列等
        tmp2 = make_hipComplex(hipCrealf(opponent[idy * sx + idx]), hipCimagf(opponent[idy * sx + idx]));

        outmlt[idy * sx + idx] = hipCmulf(tmp1, tmp2);
    }

}


void Hcudaf_shiftf(float* devReH, float* devImH, int x, int y, float d, float z, float lamda, dim3 grid, dim3 block) {
    float* ReH, * ImH;
    hipMalloc((void**)&ReH, sizeof(float) * x * y);
    hipMalloc((void**)&ImH, sizeof(float) * x * y);

    float u = 1 / (x * d), v = 1 / (y * d);

    Hcudaf << <grid, block >> > (ReH, ImH, x, y, u, v, z, lamda);
    shiftf << <grid, block >> > (devReH, devImH, ReH, ImH, x, y);

    hipFree(ReH);
    hipFree(ImH);
}

//use
void Hcudashiftcom(hipComplex* dev, int x, int y, float z, float d, float lamda, dim3 grid, dim3 block) {
    hipComplex* tmp;
    hipMalloc((void**)&tmp, sizeof(hipComplex) * x * y);

    HcudacuCom << <grid, block >> > (tmp, x, y, z, d, lamda);
    shiftCom << <grid, block >> > (dev, tmp, x, y);

    hipFree(tmp);

}


__global__ void cucompower(double* power, hipComplex* dev, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        power[idx] = sqrt((double)sqr(hipCrealf(dev[idx])) + (double)sqr(hipCimagf(dev[idx])));

    }
}

//use
__global__ void elimpadcucompower(double* power ,int sx, int sy, hipComplex* dev, int lx, int ly)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    //レジスタver
    double tmp1, tmp2;

    if (idx < sx && idy < sy) {
        tmp1 = (double)sqr(hipCrealf(dev[(idy + ly / 4) * lx + (idx + lx / 4)]));
        tmp2 = (double)sqr(hipCimagf(dev[(idy + ly / 4) * lx + (idx + lx / 4)]));
        power[idy * sx + idx] = sqrt( tmp1 + tmp2 );
    }
}



//use
__global__ void cunormaliphase(hipComplex* out, double* normali, int s)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < s) {

        out[idx] = make_hipComplex((float)cos(2 * M_PI * normali[idx]), (float)sin(2 * M_PI * normali[idx]));

    }

}


